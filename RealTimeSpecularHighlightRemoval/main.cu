#include <opencv2\opencv.hpp>
#include "SpecularHighlightRemoval.h"

int main(int argc, char **argv) 
{

	if(argc != 2) 
	{
		printf("Usage: SpecularHighlightRemoval.exe imagefile.extension\n");
		return 0;
	}

	cv::Mat inputImage = cv::imread(argv[1]);
	cv::Mat outputImage;

	SpecularHighlightRemoval specularHighlightRemoval;
	specularHighlightRemoval.initialize(inputImage.rows, inputImage.cols);
	outputImage = specularHighlightRemoval.run(inputImage);

	while(cv::waitKey(33) != 13) 
	{
		cv::imshow("Input Image", inputImage);
		cv::imshow("Output Image", outputImage);
	}

	return 0;
	
}
